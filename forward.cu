#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <thread>
#include <hipDNN.h>

#include "backprop.cuh"
#include "utils.hpp"

const int IMAGE_SIZE  = 784;
const int NUM_IMAGES  = 60000;

const int INPUT_DIM  = 784;
const int HIDDEN_DIM = 256;
const int OUTPUT_DIM = 10;

const int BATCH_SIZE = 64;

void back(int d1, int d2, float* dev_var, std::string vName){

    std::vector<float> vBack(d1*d2);
    hipMemcpy(vBack.data(), dev_var, sizeof(float)*d1*d2, hipMemcpyDeviceToHost);

    int x  = min(d1, 100);

    std::cout << "\n" << vName << " : \n";
    for (int i=0; i < d1; i++){
        for (int j=0; j < d2; j++){
            std::cout << vBack[i*d2+j] << " ";
        }
        std::cout <<"\n";
    }
}

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_KERNEL() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("Kernel error %s:%d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}
int main(){
    std::vector<float> X_train(NUM_IMAGES*IMAGE_SIZE), y_train(NUM_IMAGES);

    if (!read_mnist_data("data/train_mnist_images.bin",
                         "data/train_mnist_labels.bin",
                          X_train, 
                          y_train,
                          NUM_IMAGES,
                          IMAGE_SIZE
                        )) {
            return -1;
        }

    // first batch //
    std::vector<float> X_batch(BATCH_SIZE * INPUT_DIM);  // Batch_size (y)  x INPUT_DIM (x) >> [64, 784jj]
    std::vector<float> y_batch(BATCH_SIZE);              // Batch_size (y)  x INPUT_DIM (x) >> [64, 784]

    std::copy(X_train.begin(), X_train.begin() + BATCH_SIZE * INPUT_DIM, X_batch.begin());
    std::copy(y_train.begin(), y_train.begin() + BATCH_SIZE, y_batch.begin());

    std::vector<float> W1_h(INPUT_DIM*HIDDEN_DIM);
    std::vector<float> b1_h(HIDDEN_DIM);
    utils::xavier_init(W1_h.data(), b1_h.data(), INPUT_DIM, HIDDEN_DIM);

    std::vector<float> W2_h(HIDDEN_DIM*OUTPUT_DIM);
    std::vector<float> b2_h(OUTPUT_DIM);
    utils::xavier_init(W2_h.data(), b2_h.data(),HIDDEN_DIM, OUTPUT_DIM);


    float *X_train_d, *y_train_d;

    //forward stuf, 
    float *W1_d, *b1_d, *Y1_d, *Z1_d, *A1_d;  
    float *W2_d, *b2_d, *Y2_d, *Z2_d, *A2_d;  

    // Y1_h = X @ W1_h   >> [B, 10] >> [64x10]
    // Z1_h = Y1_h + b1_h 
    // A1_h = activation(Z1_h) // Relu then softmax.

    float* L, *l;      
    
    //TODO:
    //  drop the L eventually, go for the l.
    //  fuse X@W+b as one op. 

    hipMalloc((void **) &X_train_d, sizeof(float)*X_batch.size());
    hipMalloc((void **) &y_train_d, sizeof(float)*y_batch.size());

    // first layer
    hipMalloc((void **) &W1_d, sizeof(float)*W1_h.size());
    hipMalloc((void **) &b1_d, sizeof(float)*b1_h.size());
    hipMalloc((void **) &Y1_d, sizeof(float)*BATCH_SIZE*HIDDEN_DIM);
    hipMalloc((void **) &Z1_d, sizeof(float)*BATCH_SIZE*HIDDEN_DIM);
    hipMalloc((void **) &A1_d, sizeof(float)*BATCH_SIZE*HIDDEN_DIM);

    // second layer
    hipMalloc((void **) &W2_d, sizeof(float)*W2_h.size());
    hipMalloc((void **) &b2_d, sizeof(float)*b2_h.size());
    hipMalloc((void **) &Y2_d, sizeof(float)*BATCH_SIZE*OUTPUT_DIM);
    hipMalloc((void **) &Z2_d, sizeof(float)*BATCH_SIZE*OUTPUT_DIM);
    hipMalloc((void **) &A2_d, sizeof(float)*BATCH_SIZE*OUTPUT_DIM);

    hipMalloc((void **) &L, sizeof(float)*BATCH_SIZE);
    hipMalloc((void **) &l, sizeof(float));

    // copy weights.
    hipMemcpy(W1_d, W1_h.data(), W1_h.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b1_d, b1_h.data(), b1_h.size()*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(W2_d, W2_h.data(), W2_h.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b2_d, b2_h.data(), b2_h.size()*sizeof(float), hipMemcpyHostToDevice);

    //back(INPUT_DIM, HIDDEN_DIM, W1_d);

    // backprop stuf, 
    float *dW1_d, *db1_d, *dZ1_d, *dA1_d;
    float *dW2_d, *db2_d, *dZ2_d;// *dA2_d;

    hipMalloc((void **) &dW1_d, sizeof(float)*W1_h.size());
    hipMalloc((void **) &db1_d, sizeof(float)*b1_h.size());
    hipMalloc((void **) &dZ1_d, sizeof(float)*BATCH_SIZE*HIDDEN_DIM);
    hipMalloc((void **) &dA1_d, sizeof(float)*BATCH_SIZE*HIDDEN_DIM);

    hipMalloc((void **) &dW2_d, sizeof(float)*W2_h.size());
    hipMalloc((void **) &db2_d, sizeof(float)*b2_h.size());
    hipMalloc((void **) &dZ2_d, sizeof(float)*BATCH_SIZE*OUTPUT_DIM);
    //hipMalloc((void **) &dA2_d, sizeof(float)*BATCH_SIZE*OUTPUT_DIM);

    dim3 blockDim16(16,16);     

    dim3 gridDimHB(ceil(HIDDEN_DIM/16.0f),ceil(BATCH_SIZE/16.0f)); // 16 x 4
    dim3 gridDimOB(ceil(OUTPUT_DIM/16.0f),ceil(BATCH_SIZE/16.0f)); // 1 x 4
    dim3 gridDimOH(ceil(OUTPUT_DIM/16.0f),ceil(HIDDEN_DIM/16.0f)); //1 x 16
    dim3 gridDimHI(ceil(HIDDEN_DIM/16.0f),ceil(INPUT_DIM/16.0f)); // 16 x 49

    printf("Launch config HB: blocks(%d,%d) threads(%d,%d)\n", 
       gridDimHB.x, gridDimHB.y, blockDim16.x, blockDim16.y);

    printf("Launch config OB: blocks(%d,%d) threads(%d,%d)\n", 
       gridDimOB.x, gridDimOB.y, blockDim16.x, blockDim16.y);
    

    printf("Launch config HI: blocks(%d,%d) threads(%d,%d)\n", 
       gridDimHI.x, gridDimHI.y, blockDim16.x, blockDim16.y);

    for (unsigned int batch = 0 ; batch < 50; batch++){

        CHECK_CUDA(hipMemcpy(X_train_d, X_train.data()+batch*BATCH_SIZE*INPUT_DIM, X_batch.size()*sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(y_train_d, y_train.data()+batch*BATCH_SIZE, y_batch.size()*sizeof(float), hipMemcpyHostToDevice));

        // forward pass

        // first layer
        mult<<<gridDimHB, blockDim16>>>(X_train_d, W1_d, Y1_d, 
                        BATCH_SIZE, INPUT_DIM, HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());


        coalesced_bias<<<gridDimHB, blockDim16>>>(Z1_d, Y1_d, b1_d, HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());


        relu<<<gridDimHB, blockDim16>>>(A1_d, Z1_d, HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        // second layer
        mult<<<gridDimOB, blockDim16>>>(A1_d, W2_d, Y2_d, BATCH_SIZE, HIDDEN_DIM, OUTPUT_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        //back(2, HIDDEN_DIM, Y2_d, "Y2_d, after mult");

        coalesced_bias<<<gridDimOB, blockDim16>>>(Z2_d, Y2_d, b2_d, OUTPUT_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        //back(2, HIDDEN_DIM, Z2_d, "Z2_d, after + b2_d");

        softmax<<<gridDimOB, blockDim16>>>(A2_d, Z2_d, OUTPUT_DIM); 
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        //back(8, OUTPUT_DIM, A2_d, "softmax");

        logloss<<<ceil(BATCH_SIZE/32.), 32>>>(L, A2_d, y_train_d, OUTPUT_DIM);  
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        //back(1, 10, L, "logloss");

        rLoss<<<ceil(BATCH_SIZE/32.), 32>>>(l, L);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        back(1,1,l, "loss per batch");


        //// backward starts here: 

        dZ<<<gridDimOB,blockDim16>>>(dZ2_d, A2_d, y_train_d, OUTPUT_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        // dW2  = A1^T @ dZ2 //db2
        mult_A_T_B<<<gridDimOH, blockDim16>>>(A1_d, dZ2_d, dW2_d, HIDDEN_DIM, BATCH_SIZE, OUTPUT_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        db<<<1,OUTPUT_DIM>>>(db2_d, dZ2_d, OUTPUT_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

       // dA1 = dZ2 @ W2^T
        mult_A_B_T<<<gridDimHB, blockDim16>>>(dZ2_d, W2_d, dA1_d, BATCH_SIZE,OUTPUT_DIM,HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        dRelu<<<gridDimHB, blockDim16>>>(dA1_d, Z1_d, dZ1_d, HIDDEN_DIM);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_KERNEL();

        // dW1 = X^T @ dZ1 [I,B] @ [B, H] == [I,H] >> (16,49)
        mult_A_T_B<<<gridDimHI, blockDim16>>>(X_train_d, dZ1_d, dW1_d, INPUT_DIM, BATCH_SIZE, HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        db<<<1, HIDDEN_DIM>>>(db1_d, dZ1_d, HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        update2D<<<gridDimHI,blockDim16>>>(W1_d, dW1_d, INPUT_DIM,  HIDDEN_DIM);
        update1D<<<1,HIDDEN_DIM>>>(b1_d, db1_d, HIDDEN_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        update2D<<<gridDimOH,blockDim16>>>(W2_d, dW2_d, HIDDEN_DIM, OUTPUT_DIM);
        update1D<<<1,INPUT_DIM>>>(b2_d, db2_d, OUTPUT_DIM);
        CHECK_KERNEL();
        CHECK_CUDA(hipDeviceSynchronize());

        //hipError_t err = hipGetLastError();
        ////if (err != hipSuccess) {
        ////    std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << "\n";
        ////    return;
        ////}



        //back(INPUT_DIM, HIDDEN_DIM, dW1_d, "dW1: ");
        //back(1, HIDDEN_DIM, b1_d, "b1: ");

    }

    return 0;
}
