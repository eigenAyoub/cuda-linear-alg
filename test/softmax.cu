#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <thread>
#include <hipDNN.h>
#include <unistd.h> 
#include <iomanip>  

#include "../utils.hpp"

const int BATCH_SIZE = 2048;
const int OUTPUT_DIM = 1024;

const int BLOCKDIMY  = 16;
const int BLOCKDONE  = 1;


__global__ void softmaxShared(float* A, float *Z, int hidden_dim){

    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    __shared__ float buffPerBlock[1];
    __shared__ float max[1];


    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDONE) {
        max[0] = Z[row*hidden_dim];
        for (int i = 1; i < hidden_dim; i++){
            max[0] = fmaxf(max[threadIdx.y],Z[row*hidden_dim+i]) ;
        }
        buffPerBlock[0] = 0.0f;
    }
    __syncthreads();

    // maxP
    if (col < hidden_dim && row < BATCH_SIZE) {
        A[row*hidden_dim+col] = exp(Z[row*hidden_dim+col] -  max[threadIdx.y]);
    }
    __syncthreads();
    
    // now Z is useles; I can use it sum over? using a simple reduction scheme?
    // maybe Z is being used eventually.
    // you don't want to keep changing things
    // you stupid fuck
    
    if (col == 0 && row < BATCH_SIZE) {
        buffPerBlock[0] = A[row*hidden_dim];
        for (int i = 1; i < hidden_dim; i++){
            buffPerBlock[0] += A[row*hidden_dim+i];
        }
    }
    __syncthreads();

    // maxP
    if (col < hidden_dim && row < BATCH_SIZE) {
        A[row*hidden_dim+col] /= buffPerBlock[threadIdx.y];
    }
}

// block of 16 x 16
// ok
__global__ void softmaxTiled(float* A, float *Z, int hidden_dim){

    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    extern __shared__ float buff[];

	// compute max
    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDIMY) {
        buff[threadIdx.y] = Z[row*hidden_dim];
        float zt = 0.0f;
        for (int i = 1; i < hidden_dim; i++){
            zt = Z[row*hidden_dim+i];
            buff[threadIdx.y] = fmaxf(buff[threadIdx.y],zt);
        }
    }
    __syncthreads();

	// update A: tiled way
    if (col < blockDim.x){ // same shared memory
        for (unsigned int tile = 0 ; tile+col < hidden_dim; tile+=blockDim.x){
			A[row*hidden_dim+col+tile] = exp(Z[row*hidden_dim+col+tile] -  buff[threadIdx.y]);
        }
        //if (col == 0)   buff[threadIdx.y] = 0.0f; // reset for sum
    }
    __syncthreads();

    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDIMY) {
		buff[threadIdx.y] = A[row*hidden_dim];
        for (int i = 1; i < hidden_dim; i++){
			buff[threadIdx.y] += A[row*hidden_dim+i];
        }
    }
    __syncthreads();

    if (col < blockDim.x){ 
        for (unsigned int tile = 0 ; tile+col < hidden_dim; tile+=blockDim.x){
			A[row*hidden_dim+col+tile] /= buff[threadIdx.y];
        }
    }
}

__global__ void softmax(float* A, float *Z, int hidden_dim){

    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    __shared__ float buffPerBlock[BLOCKDIMY];
    __shared__ float max[BLOCKDIMY];

    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDIMY) {
        max[threadIdx.y] = Z[row*hidden_dim];
        for (int i = 1; i < hidden_dim; i++){
            float zt = Z[row*hidden_dim+i];
            max[threadIdx.y] = fmaxf(max[threadIdx.y],zt) ;
        }
        buffPerBlock[threadIdx.y] = 0.0f;
    }
    __syncthreads();

    if (col == 0 && row < BATCH_SIZE) {
        float sum = 0.0f;
        for (int i = 0; i < hidden_dim; i++){
            float At = exp(Z[row*hidden_dim+i] -  max[threadIdx.y]);
            A[row*hidden_dim+i] = At;
            sum += At;
        }
        buffPerBlock[threadIdx.y] = sum;
    }
    __syncthreads();

    if (col == 0 && row < BATCH_SIZE) {
        for (int i = 0; i < hidden_dim; i++){
            A[row*hidden_dim+i] = A[row*hidden_dim+i]/buffPerBlock[threadIdx.y];
        }
    }
}

void cpusmx(float *A, float *Z, int outDim, int B = BATCH_SIZE){
    for (int i = 0; i < B; ++i) {
        float maxVal = Z[i * outDim + 0];
        for (int j = 1; j < outDim; ++j) {
            if(Z[i*outDim+j] > maxVal) {
                maxVal = Z[i*outDim+j];
            }
        }
        float sumExp = 0.f;
        for (int j = 0; j < outDim; ++j) {
            float e = std::exp(Z[i * outDim + j] - maxVal);
            A[i * outDim + j] = e;
            sumExp += e;
        }
        for (int j = 0; j < outDim; ++j) {
            A[i * outDim + j] /= sumExp;
        }
    }
}

void back(int d1, int d2, float* dev_var, std::string vName){

    std::vector<float> vBack(d1*d2);
    hipMemcpy(vBack.data(), dev_var, sizeof(float)*d1*d2, hipMemcpyDeviceToHost);

    int x  = min(d1, 100);

    std::cout << "\n" << vName << " : \n";
    for (int i=0; i < d1; i++){
        for (int j=0; j < d2; j++){
            std::cout << vBack[i*d2+j] << " ";
        }
        std::cout <<"\n";
    }
    std::cout <<"\n";
}

void visual(int d1, int d2, float* dev_var, std::string vName){

    std::cout << "\n" << vName << " : \n";
    for (int i=0; i < d1; i++){
        for (int j=0; j < d2; j++){
            std::cout << dev_var[i*d2+j] << " ";
        }
        std::cout <<"\n";
    }
    std::cout <<"\n";
}


__global__ void softmaxWithDRAMBuffer(float* A, float *Z, int hidden_dim){

    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDONE) {
        A[(row+1)*(hidden_dim)-1] = Z[row*(hidden_dim-2)];
        for (int i = 1; i < (hidden_dim-2); i++){
            A[(row+1)*(hidden_dim)-1] = fmaxf(A[(row+1)*(hidden_dim)-1], Z[row*(hidden_dim-2)+i]) ;
        }
        A[(row+1)*(hidden_dim)-2] = 0.0f; // sum buffer.
    }
    __syncthreads();

    if (col == 0 && row < BATCH_SIZE) {
        float sum = 0.0f;
        for (int i = 0; i < (hidden_dim-2); i++){
            float At = exp(Z[row*(hidden_dim-2)+i] - A[(row+1)*hidden_dim-1]); // At = exp(Z - maxPerRow)
            A[row*hidden_dim+i] = At;
            sum += At;
        }  
        A[(row+1)*(hidden_dim)-2] = sum;
    }
    __syncthreads();

    if (col == 0 && row < BATCH_SIZE) {
        for (int i = 0; i < (hidden_dim-2); i++){
            A[row*hidden_dim+i] = A[row*hidden_dim+i]/A[(row+1)*(hidden_dim)-2];
        }
    }
}

__global__ void softmaxWithDRAMBuffer2(float* A, float *Z, int hidden_dim){

    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDONE) {
        A[(row+1)*(hidden_dim)-1] = Z[row*(hidden_dim-2)];
        for (int i = 1; i < (hidden_dim-2); i++){
            A[(row+1)*(hidden_dim)-1] = fmaxf(A[(row+1)*(hidden_dim)-1], Z[row*(hidden_dim-2)+i]) ;
        }
        A[(row+1)*(hidden_dim)-2] = 0.0f; // sum buffer.
    }
    __syncthreads();

    if (row < BATCH_SIZE && col < hidden_dim-2) {
        A[row*hidden_dim+col] = exp(Z[row*(hidden_dim-2)+col] - A[(row+1)*hidden_dim-1]); // At = exp(Z - maxPerRow)
    }
    __syncthreads();

    if (col == 0 && row < BATCH_SIZE) {
        float sum = 0.0f;
        for (int i = 0; i < (hidden_dim-2); i++){
            sum += A[row*hidden_dim+i] ;
        }  
        A[(row+1)*(hidden_dim)-2] = sum;
    }
    __syncthreads();


    if (row < BATCH_SIZE && col < hidden_dim-2) {
        A[row*hidden_dim+col] /= A[(row+1)*hidden_dim-2]; // At /= maxPerRow 
    }
}


bool verifyMaxIndices(const float* A, const float* B, 
                     int batch_size, int output_dim) {
    for (int i = 0; i < batch_size; i++) {
        // Find max indices
        int max_idx1 = 0, max_idx2 = 0;
        float max1 = A[i * output_dim];
        float max2 = B[i * output_dim];
        
        for (int j = 1; j < output_dim; j++) {
            if (A[i * output_dim + j] > max1) {
                max1 = A[i * output_dim + j];
                max_idx1 = j;
            }
            if (B[i * output_dim + j] > max2) {
                max2 = B[i * output_dim + j];
                max_idx2 = j;
            }
        }
        
        if (max_idx1 != max_idx2) {
            printf("Row %d: Different max indices: %d vs %d\n", 
                   i, max_idx1, max_idx2);
            return false;
        }
    }
    return true;
}


__global__ void softmaxWithDRAMBufferAtomic(float* A, float *Z, int hidden_dim){

    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    if (col == 0 && row < BATCH_SIZE && threadIdx.y < BLOCKDONE) {
        float maxVal = Z[row*(hidden_dim-2)];
        for (int i = 1; i < (hidden_dim-2); i++){
            maxVal = fmaxf(maxVal, Z[row*(hidden_dim-2)+i]) ;
        }
        A[(row+1)*(hidden_dim)-1] = maxVal;
        //A[(row+1)*(hidden_dim)-2] = 0.0f; 
    }
    __syncthreads();

    if (row < BATCH_SIZE && col < hidden_dim-2) {
        float v = exp(Z[row*(hidden_dim-2)+col] - A[(row+1)*hidden_dim-1]); // At = exp(Z - maxPerRow)
        A[row*hidden_dim+col] = v ;
        atomicAdd(&A[(row+1)*hidden_dim - 2], v);
    }
    __syncthreads();

    if (row < BATCH_SIZE && col < hidden_dim-2) {
        A[row*hidden_dim+col] /= A[(row+1)*hidden_dim-2]; // At /= maxPerRow 
    }
}

float returnMaxDiff(float * A, float * B, int bs, int dA, int d){

    // bs: batch size :p
    // A gpu matrix
    // dA width of A
    // d = dA - dB

    float diff = 1e-30f;
    float tempDiff = 0.0f;

    for (unsigned int r  = 0; r < bs; r++){
        for (unsigned int c = 0; c  < dA-d; c++){
            tempDiff = std::abs(A[r*dA+c]-B[r*(dA-d)+c]);
            if (tempDiff > diff)    diff = tempDiff;
        }
    }
    return diff;
}


void gpuSoftmax(float* data, int batch_size, int hidden_dim) {
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Allocate device memory
    float *d_data;
    size_t matrix_size = batch_size * hidden_dim * sizeof(float);
    hipMalloc((void**)&d_data, matrix_size);
    hipMemcpy(d_data, data, matrix_size, hipMemcpyHostToDevice);

    // Create tensor descriptor for batch_size x hidden_dim matrix
    hipdnnTensorDescriptor_t data_desc;
    hipdnnCreateTensorDescriptor(&data_desc);
    // NCHW: batch_size x hidden_dim x 1 x 1
    hipdnnSetTensor4dDescriptor(
        data_desc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batch_size,    // N: number of images
        hidden_dim,    // C: number of channels (features)
        1,            // H: height
        1             // W: width
    );

    // Perform softmax
    float alpha = 1.0f, beta = 0.0f;
    hipdnnSoftmaxForward(
        cudnn,
        //HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_FAST,
        HIPDNN_SOFTMAX_MODE_CHANNEL,  // Softmax across hidden_dim
        &alpha,
        data_desc,
        d_data,
        &beta,
        data_desc,
        d_data
    );

    // Copy result back
    hipMemcpy(data, d_data, matrix_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_data);
    hipdnnDestroyTensorDescriptor(data_desc);
    hipdnnDestroy(cudnn);
}



int main(){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::vector<float> W(BATCH_SIZE*OUTPUT_DIM);   // 64 x 32
    std::vector<float> b(OUTPUT_DIM);
    utils::xavier_init(W.data(), b.data(),BATCH_SIZE, OUTPUT_DIM);

    std::vector<float> W_cudnn = W; // atomic

    std::vector<float> W_cpu(BATCH_SIZE*OUTPUT_DIM, 0.0f);

    float ms;
    hipEventRecord(start);

    gpuSoftmax(W_cudnn.data(), BATCH_SIZE, OUTPUT_DIM);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << std::left << std::setw(30) <<">> cuDNN implementation took" 
                                            << utils::formatTime(ms) << std::endl;




    // CPU time.
    utils::Timer timeCPU = utils::Timer(">> CPU - ");
    cpusmx(W_cpu.data(), W.data(), OUTPUT_DIM);
    std::string cpuTime = timeCPU.report();


    std::cout << "CPU and cudnn implementation difference: "<< returnMaxDiff(W_cpu.data(),W_cudnn.data(), BATCH_SIZE, OUTPUT_DIM, 0)<< "\n";
    float epsilon = 1e-4f;  // Adjust based on precision needs

    // Check each row
    for (int i = 0; i < BATCH_SIZE; i++) {
        // Find max values for each implementation
        float max1 = W_cpu[i * OUTPUT_DIM];
        float max2 = W_cudnn[i * OUTPUT_DIM];
        
        for (int j = 1; j < OUTPUT_DIM; j++) {
            max1 = fmaxf(max1, W_cpu[i * OUTPUT_DIM + j]);
            max2 = fmaxf(max2, W_cudnn[i * OUTPUT_DIM + j]);
        }
        
        // Compare max values
        float max_diff = fabsf(max1 - max2);
        if (max_diff > epsilon) {
            printf("Row %d: Max difference too large: %e\n", i, max_diff);
        }

        // Verify row sums to 1
        float sum1 = 0.0f, sum2 = 0.0f;
        float max_element_diff = 0.0f;
        
        for (int j = 0; j < OUTPUT_DIM; j++) {
            sum1 += W_cpu[i * OUTPUT_DIM + j];
            sum2 += W_cudnn[i * OUTPUT_DIM + j];
            max_element_diff = fmaxf(max_element_diff, 
                                    fabsf(W_cpu[i * OUTPUT_DIM + j] - W_cudnn[i * OUTPUT_DIM + j]));
        }
        
        if (fabsf(sum1 - 1.0f) > epsilon || fabsf(sum2 - 1.0f) > epsilon) {
            printf("Row %d: Sum not close to 1.0: %f, %f\n", i, sum1, sum2);
        }
        
        if (max_element_diff > epsilon) {
            printf("Row %d: Max element difference: %e\n", i, max_element_diff);
        }
    }

    std::cout << "Reality " << verifyMaxIndices(W_cpu.data(), W_cudnn.data(), BATCH_SIZE, OUTPUT_DIM);

    // it's just GPU now.

    std::vector<float> W_gpu(BATCH_SIZE*OUTPUT_DIM, 0.0f);
    std::vector<float> W_gpu2(BATCH_SIZE*(OUTPUT_DIM+2), 0.0f);
    std::vector<float> W_gpu3(BATCH_SIZE*(OUTPUT_DIM+2), 0.0f);
    std::vector<float> W_gpu4(BATCH_SIZE*(OUTPUT_DIM+2), 0.0f); // atomic

    std::vector<float> W_gpu5(BATCH_SIZE*(OUTPUT_DIM), 0.0f); // atomic

    float btoMB = 1024.0f*1024.0f;
    std::cout << "\nMem size for matrix: "  << BATCH_SIZE*OUTPUT_DIM*sizeof(float)/btoMB << " MB\n";
    std::cout << "Mem size for matrix w/ buff: "  << BATCH_SIZE*(OUTPUT_DIM+2)*sizeof(float)/btoMB << "MB\n";

    float *W_d;
    float *A_d;  // we bringing this back.
    float *A_d2; // we bringing this back.
    float *A_d3; // we bringing this back.
    float *A_d4; // we bringing this back.
    float *A_d5; // we bringing this back.

    hipMalloc((void **) &W_d,  sizeof(float)*W.size());
    hipMalloc((void **) &A_d,  sizeof(float)*W.size());
    hipMalloc((void **) &A_d2, sizeof(float)*W_gpu2.size());  // [BATCH_SIZE x OUTPUT_DIM+2]
    hipMalloc((void **) &A_d3, sizeof(float)*W_gpu3.size());  // [BATCH_SIZE x OUTPUT_DIM+2]
    hipMalloc((void **) &A_d4, sizeof(float)*W_gpu4.size());  // [BATCH_SIZE x OUTPUT_DIM+2]
    hipMalloc((void **) &A_d5, sizeof(float)*W_gpu5.size());  // [BATCH_SIZE x OUTPUT_DIM+2]

    hipMemset(A_d4, 0,sizeof(float)*W_gpu4.size());


    hipEventRecord(start);           

    hipMemcpy(W_d, W.data(), sizeof(float)*W.size(), hipMemcpyHostToDevice);

    hipEventRecord(stop);           
    hipEventSynchronize(stop);      
    hipEventElapsedTime(&ms, start, stop);
    std::cout << std::left << std::setw(30) << "Time to hipMemcpy: " 
                                            << utils::formatTime(ms) << std::endl;
    std::cout << "\n";



    float out = 64.0f;
    float batch = 4.0f;

    dim3 blockDim16(out,batch);     
    dim3 gridDimOB(ceil(OUTPUT_DIM/out),ceil(BATCH_SIZE/batch)); // 2 x 4
    
    // warm-up.

    for (int i = 0; i < 20; i++){
        softmax<<<gridDimOB, blockDim16>>>(A_d, W_d, OUTPUT_DIM); 
    }
    hipDeviceSynchronize();

    // cpu time:
    std::cout << "Time of different implementations:\n\n";
    std::cout << timeCPU.report()  << std::endl;

    // GPU - CUDA, Naive
    hipEventRecord(start);

    softmaxTiled<<<gridDimOB, blockDim16, batch*sizeof(float)>>>(A_d, W_d, OUTPUT_DIM); 
    // 285 ms > normal

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << std::left << std::setw(30) <<">> GPU - naive softmax: " 
                                            << utils::formatTime(ms) << std::endl;
    hipMemcpy(W_gpu.data(), A_d, sizeof(float)*W.size(), hipMemcpyDeviceToHost);



    // GPU - CUDA, shared; one block per row 

    hipEventRecord(start);
    dim3 blockDimO(OUTPUT_DIM,1);     
    dim3 gridDimO(1, BATCH_SIZE); 

    softmaxShared<<<gridDimO, blockDimO>>>(A_d5, W_d, OUTPUT_DIM); 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << std::left << std::setw(30) <<">> GPU - softmax sharedmem: " 
                                            << utils::formatTime(ms) << std::endl;
    hipMemcpy(W_gpu5.data(), A_d5, sizeof(float)*W.size(), hipMemcpyDeviceToHost);


    
    // GPU - CUDA, Naive, with buffer in DRAM, and no shared memory usage.
    dim3 blockDim11(OUTPUT_DIM,1);     
    dim3 gridDim11(1,BATCH_SIZE);     

    hipEventRecord(start);

    softmaxWithDRAMBuffer<<<gridDim11, blockDim11>>>(A_d2, W_d, OUTPUT_DIM+2); 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    std::cout << std::left << std::setw(30) 
                            << ">> GPU - softmax with buffer: " 
                            << utils::formatTime(ms) << std::endl;

    hipMemcpy(W_gpu2.data(), A_d2, sizeof(float)*W_gpu2.size(), hipMemcpyDeviceToHost);


    // GPU - CUDA, Naive, with buffer in DRAM, and no shared memory usage // slightly more intel.
    hipEventRecord(start);

    softmaxWithDRAMBuffer2<<<gridDim11, blockDim11>>>(A_d3, W_d, OUTPUT_DIM+2); 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    std::cout << std::left << std::setw(30) << ">> GPU - softmax with buffer+ " 
                                            << utils::formatTime(ms) << std::endl;
    hipMemcpy(W_gpu3.data(), A_d3, sizeof(float)*W_gpu3.size(), hipMemcpyDeviceToHost);

    // GPU - Atomic DRAM buffer 
    hipEventRecord(start);

    softmaxWithDRAMBufferAtomic<<<gridDim11, blockDim11>>>(A_d4, W_d, OUTPUT_DIM+2); 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    std::cout << std::left << std::setw(30) << ">> GPU - softmax atomic buffer+ " 
                                            << utils::formatTime(ms) << std::endl;
    hipMemcpy(W_gpu4.data(), A_d4, sizeof(float)*W_gpu4.size(), hipMemcpyDeviceToHost);


    std::cout << "\nMax of absolute distance:\n";
    /// Computing the diffs per implementation.

    float diff = 0.0f;

    diff = returnMaxDiff(W_gpu.data(), W_cpu.data(), BATCH_SIZE, OUTPUT_DIM, 0);
    std::cout << "> max(abs(CPU - GPU_NAIVE)) = " << diff << "\n";

    diff = returnMaxDiff(W_gpu5.data(), W_cpu.data(), BATCH_SIZE, OUTPUT_DIM, 0);
    std::cout << "> max(abs(CPU - GPU_shared) = " << diff << "\n";

    diff = returnMaxDiff(W_gpu2.data(), W_cpu.data(), BATCH_SIZE, OUTPUT_DIM+2, 2);
    std::cout << "> max(abs(CPU - GPU_Buff) = " << diff << "\n";

    diff = returnMaxDiff(W_gpu3.data(), W_cpu.data(), BATCH_SIZE, OUTPUT_DIM+2, 2);
    std::cout << "> max(abs(CPU - GPU_Buff_plus) = " << diff << "\n";

    diff = returnMaxDiff(W_gpu4.data(), W_cpu.data(), BATCH_SIZE, OUTPUT_DIM+2, 2);
    std::cout << "> max(abs(CPU - GPU_atomic) = " << diff << "\n";

}
