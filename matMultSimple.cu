#include <iostream>
#include <hip/hip_runtime.h>


__global__ void matMul(float *A, float *B, float *C, int N){
    int col = 0;
    int row = 0;
    C[row*N + col] = 0;
}


int main(){
    int N = 256*5;

    float A[N][N];
    float B[N][N];
    float C[N][N];

    int size = N*N*sizeof(float);
    
    // malloc
    float *A_d, *B_d, *C_d;

    hipMalloc((void **) &A_d,size);
    hipMalloc((void **) &B_d,size);
    hipMalloc((void **) &C_d,size);

    // Memcpy

    hipMemcpy(A_d,A,size,hipMemcpyHostToDevice);
    hipMemcpy(B_d,B,size,hipMemcpyHostToDevice);
    
    // computations:
    int numOfBlocks = 5;
    int threadsPerBlock = 256;

    matMul<<<numOfBlocks, threadsPerBlock>>>(A_d, B_d, C_d, N);

    hipMemcpy(C,C_d,size,hipMemcpyDeviceToHost);

    // moving data back


    // free Cuda


    return 0;
}